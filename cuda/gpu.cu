
#include <hip/hip_runtime.h>

__global__ void cuAutoAdjust_dev(unsigned char *s, unsigned char *d, int count, int *autoTable, bool invert)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < count)
  {
    int val = s[i];

    val = autoTable[val];

    if (invert)
    {
      val = 255 - val;
    }

    d[i] = val;
  }
}

__global__ void cuProcessImage_dev(unsigned char *s, unsigned char *d, int count, int *contrastTable, int *gammaTable, bool invert, float brightness)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < count)
  {
    int val = s[i];

    val += brightness;
    if (val < 0)
    {
      val = 0;
    }
    else
    if( val > 255 )
    {
      val = 255;
    }

    val = contrastTable[val];
    val = gammaTable[val];

    val = invert ? 255 - val : val;

    d[i] = val;
  }
}

extern "C" void cuProcessImage(unsigned char *s, unsigned char *d, int count, int *contrastTable, int *gammaTable, bool invert, float brightness)
{
  int threadsPerBlock = 512;
  int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

  cuProcessImage_dev<<<blocksPerGrid, threadsPerBlock>>>(s, d, count, contrastTable, gammaTable, invert, brightness);
}


extern "C" void cuAutoAdjust(unsigned char *s, unsigned char *d, int count, int *autoTable, bool invert)
{
  int threadsPerBlock = 512;
  int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

  cuAutoAdjust_dev<<<blocksPerGrid, threadsPerBlock>>>(s, d, count, autoTable, invert);
}
